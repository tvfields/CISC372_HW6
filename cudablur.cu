#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define BSIZE = 256;

__global__ void computeRow(float *src, float *dest, int pWidth, int radius, int bpp, int height){
    int i; 

    int bradius = radius * bpp;
     
    int row = threadIdx.x + (blockIdx.x * blockDim.x);

    if (height > row){
        for (i = 0; i < bpp; i++){
            dest[row * pWidth + i] = src[row * pWidth + i];
        }

        for (i = bpp; i < bradius * 2 * bpp; i++){
            dest[row * pWidth + i] = src[row * pWidth + i] + dest[row * pWidth + i - bpp];
        }

        for (i = bradius * 2 + bpp; i < pWidth; i++){
            dest[row * pWidth + i] = src[row * pWidth + i] + dest[row * pWidth + i - bpp] - src[row * pWidth + i - 2 * bradius - bpp];
        }

        for (i = bradius; i < pWidth; i++){
            dest[row * pWidth + i - bradius] = dest[row * pWidth + i] / (radius * 2 + 1);
        }
   
        for (i = 0; i < bradius; i++){
            dest[row * pWidth + i] = 0;
            dest[(row + 1) * pWidth - 1 - i] = 0;
        }
    }
}

__global__ void computeColumn(uint8_t *src, float *dest, int pWidth, int height, int radius, int bpp){
    int i;
    int col = threadIdx.x + (blockIdx.x * blockDim.x);

    if (pWidth > col){
        dest[col] = src[col];

        for (i = 1; i <= radius * 2; i++){
            dest[i * pWidth + col] = src[i * pWidth + col] + dest[(i - 1) * pWidth + col];
        }
        
        for (i = radius * 2 + 1; i < height; i++){
            dest[i * pWidth + col] = src[i * pWidth + col] + dest[(i - 1) * pWidth + col] - src[(i - 2 * radius - 1) * pWidth + col];
        }

        for (i = radius; i < height; i++){
            dest[(i - radius) * pWidth + col] = dest[i * pWidth + col] / (radius * 2 + 1);
        }

        for (i = 0; i < radius; i++)
        {
            dest[i * pWidth + col] = 0;
            dest[(height - 1) * pWidth - i * pWidth + col] = 0;
        }
    }
}

int Usage(char *name)
{
    printf("%s: <filename> <blur radius>\n\tblur radius=pixels to average on any side of the current pixel\n", name);
    return -1;
}

int main(int argc, char **argv){
    long t1, t2;
    int r = 0;
    int i;
    int w, h, bpp, pw;
    char *fname;
    uint8_t *img;
    float *dest, *mid;
    uint8_t *dest_img;

    if (argc != 3){
        return Usage(argv[0]);
    }

    fname = argv[1];
    sscanf(argv[2], "%d", &r);
    img = stbi_load(filename, &w, &h, &bpp, 0);

    pw = w * bpp;

    hipMalloc(&dest_img, sizeof(uint8_t) * pw * h);
    hipMallocManaged(&mid, sizeof(float) * pw * h);
    hipMallocManaged(&dest, sizeof(float) * pw * h);

    int bnum = (pw + (BSIZE - 1)) / BSIZE;

    t1 = clock();
    computeColumn<<<bnum, BSIZE>>>(dest_img, mid, pw, h, r, bpp);
    hipDeviceSynchronize();
    stbi_image_free(img); 

    bnum = (h + (BS - 1)) / BSIZE;

    computeRow<<<bnum, BSIZE>>>(mid, dest, pw, r, bpp, h);
    hipDeviceSynchronize();
    t2 = clock();
    hipFree(mid); 
    img = (uint8_t*)malloc(sizeof(uint8_t) * pw * h);
    
    for (i = 0; i < pw * h; i++){
        img[i] = (uint8_t)dest[i];
    }
    
    stbi_write_png("output.png", w, h, bpp, img, bpp * w);
    hipFree(img);
    hipFree(mid);
    hipFree(dest);
    free(img);
    printf("Blur with radius %d complete in %f seconds\n", r, (double)(t2 - t1) / (double)CLOCKS_PER_SEC);
}
